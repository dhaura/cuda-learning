#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


void check_error(hipError_t err) {
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__global__ void vec_add(const float *a, const float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main(void) {

    float *h_a, *h_b, *h_c; // Host pointers.
    float *d_a, *d_b, *d_c; // Device pointers.
    int n = 1024;
    size_t size = n * sizeof(float);

    // Allocate memory on the host.
    h_a = (float *)malloc(size);
    h_b = (float *)malloc(size);
    h_c = (float *)malloc(size);

    // Initialize host arrays.
    for (int i = 0; i < n; i++) {
        h_a[i] = i * 1.0f;
        h_b[i] = i * 2.0f;
    }

    // Allocate memory on the device.
    check_error(hipMalloc((void **)&d_a, size));
    check_error(hipMalloc((void **)&d_b, size));
    check_error(hipMalloc((void **)&d_c, size));

    // Copy data from host to device.
    check_error(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
    check_error(hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice));

    // Launch the kernel.
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    vec_add<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);
    
    check_error(hipGetLastError());
    check_error(hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost));

    // Verify the result.
    for (int i = 0; i < n; i++) {
        if (h_c[i] != h_a[i] + h_b[i]) {
            printf("Error at index %d: %f + %f != %f\n", i, h_a[i], h_b[i], h_c[i]);
        }
    }

    printf("Vector addition completed successfully.\n");

    // Free memory.
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
